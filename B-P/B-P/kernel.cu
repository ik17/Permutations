#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <set>
#include <string>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <random>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort = true)
{
	if (code != 0) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (Abort) return;
	}
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }


__device__ int factorial(int n) {
	if (n == 1) {
		return 1;
	}
	return n * factorial(n - 1);
}



__global__ void permute_kernel(char* d_A, int size) {
	int jednoD = blockIdx.x;
	int dvoD = jednoD + blockIdx.y*gridDim.x;
	int troD = dvoD + gridDim.x*gridDim.y*blockIdx.z;
	int tid;
	tid = troD * blockDim.x + threadIdx.x;
	int fakt = factorial(size);
	if (tid < fakt) {

		int* counter = new int[size];
		char* kopija = new char[size];

		for (int i = 0; i < size; i++) {
			counter[i] = 0;
			kopija[i] = d_A[i];
		}
		int j = 2;
		int temp = tid;
		for (int i = 1; i < size; i++) {
			while (temp >= (fakt / j)) {
				counter[i]++;
				temp -= fakt / j;
			}
			fakt = fakt / j;
			j++;
		}


		for (int i = 0; i < size; i++) {
			int poz = i - counter[i];
			if (poz < i) {
				char temp = kopija[i];
				kopija[i] = kopija[poz];
				kopija[poz] = temp;
			}
		}

		printf("GPU Thread: %i Permutacija: %s\n", tid, kopija);

		delete[] counter;
		delete[] kopija;

	}
}


int factorialHost(int n) {
	if (n == 1) {
		return 1;
	}
	return n * factorialHost(n - 1);
}






void funkcija(FILE *fp, int n, double *sum, double *maxi, double *mini) {
	clock_t begin = clock();
	char h_a[] = "ABCDEF";

	char* d_a;
	int duzina = 6;
	hipMalloc((void**)&d_a, sizeof(h_a));
	GPUerrchk(hipMemcpy(d_a, h_a, sizeof(h_a), hipMemcpyHostToDevice));

	int fakt = factorialHost(duzina);
	int threadNoMC = fakt; 
	char* h_svePermutacije = new char[threadNoMC * duzina];

	char* svePermutacije;
	hipMalloc((void**)&svePermutacije, sizeof(char)* threadNoMC * duzina);
	hipMemset(svePermutacije, '0', sizeof(char) * threadNoMC * duzina);



	std::set<std::string> unikatno;
	printf("\n\n B-P\n");
	int number = 1;
	while (threadNoMC / number > 320) number++;
	while (1.0*threadNoMC / number - int(threadNoMC / number) > 0) number++;
	int a = threadNoMC / number;
	permute_kernel << <number, a >> > (d_a, duzina);
	for (std::string s : unikatno) {
		std::cout << s << std::endl;
	}
	GPUerrchk(hipPeekAtLastError());
	GPUerrchk(hipDeviceSynchronize());
	time_t end = clock();
	printf("Vrijeme izvrsenja u sekundama je: %f\n", (double)(end - begin) / CLOCKS_PER_SEC);
	if (n != 0) {
		fprintf(fp, "%d,%f\n", n, (double)(end - begin) / CLOCKS_PER_SEC);
		*sum += (double)(end - begin) / CLOCKS_PER_SEC;
		if (*maxi < (double)(end - begin) / CLOCKS_PER_SEC) *maxi = (double)(end - begin) / CLOCKS_PER_SEC;
		if (*mini > (double)(end - begin) / CLOCKS_PER_SEC) *mini = (double)(end - begin) / CLOCKS_PER_SEC;
	}
}


int main()
{
	srand(time(NULL));
	FILE *fp;
	fp = fopen("C:\\Users\\ismar\\Desktop\\BP.csv", "w");
	double sum = 0.0;
	double maxi = -999999.9;
	double mini = 999999.9;
	for (int i = 0; i <= 100; i++) {
		if (fp == NULL) {
			printf("Couldn't open file\n");
			return;
		}
		funkcija(fp, i, &sum, &maxi, &mini);
	}
	fprintf(fp, "%s,%f\n", "Minimum", mini);
	fprintf(fp, "%s,%f\n", "Maximum", maxi);
	fprintf(fp, "%s,%f\n", "Prosjek", 1.0*sum / 100);
	printf("Prosjecno vrijeme izvrsavanja je: %f", 1.0*sum / 100);
	fclose(fp);
	return 0;
}
